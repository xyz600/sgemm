#include <cassert>
#include <hip/hip_runtime.h>

#include "matrix_cpu.hpp"
#include "matrix_gpu.cuh"
#include "sgemm_kernel.cuh"

#define CHECK(call)                                                                                                    \
    {                                                                                                                  \
        const hipError_t error = call;                                                                                \
        if (error != hipSuccess)                                                                                      \
        {                                                                                                              \
            printf("Error: %s:%d,  ", __FILE__, __LINE__);                                                             \
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error));                                         \
            exit(1);                                                                                                   \
        }                                                                                                              \
    }

MatrixGPU::MatrixGPU(std::size_t size)
    : size_(size)
    , stride_(exponential_ceil(size_))
{
    CHECK(hipMalloc((void**)&data_, sizeof(float) * size_ * stride_));
}

MatrixGPU::~MatrixGPU() { hipFree(data_); }

std::size_t MatrixGPU::exponential_ceil(const std::size_t size) const noexcept { return (size + size - 1) / 2 * 2; }

void MatrixGPU::multiply(const MatrixGPU& right, MatrixGPU& out) const noexcept
{
    assert(size_ == right.size_);
    assert(size_ == out.size_);

    sgemm<<<size_, size_>>>(data_, right.data_, out.data_, size_, stride_);
}

void MatrixGPU::copy_from(const MatrixCPU& src) noexcept
{
    CHECK(hipMemcpy((void*)data_, (void*)src.data(), sizeof(float) * stride_ * size_, hipMemcpyHostToDevice));
}

void MatrixGPU::copy_to(MatrixCPU& dst) const noexcept
{
    CHECK(hipMemcpy((void*)dst.data(), (void*)data_, sizeof(float) * stride_ * size_, hipMemcpyDeviceToHost));
}

void MatrixGPU::clear() noexcept { fill<<<1024, 1024>>>(data_, size_, 0); }

std::size_t MatrixGPU::size() const noexcept { return size_; }