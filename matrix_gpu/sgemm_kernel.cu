#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include "sgemm_kernel.cuh"

__global__ void fill(float* data, const int size, const float value)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < size)
    {
        data[idx] = value;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void sgemm(const float* a, const float* b, float* result, const int size, const int stride)
{
    const int thread_idx = threadIdx.y * warpSize + threadIdx.x;
    const int num_thread = blockDim.x * blockDim.y;

    constexpr int small_block_size = 2;
    constexpr int block_size_x = 64;

    // thread 単位で small_balock_size^2 だけ要素を持っている時に確保できる block_size_y
    const int block_size_y = num_thread * small_block_size * small_block_size / block_size_x;
    assert(block_size_y <= block_size_x);

    for (int i = blockIdx.y * block_size_y; i < size; i += gridDim.y * block_size_y)
    {
        for (int j = blockIdx.x * block_size_x; j < size; j += gridDim.x * block_size_x)
        {
            const int base_i = i + threadIdx.y * 2;
            const int base_j = j + threadIdx.x * 2;
            const bool has_result = (base_i < size && base_j < size);

            // 単一スレッドの結果保存用
            float local_result[small_block_size][small_block_size];
            for (int ii = 0; ii < small_block_size; ii++)
            {
                for (int jj = 0; jj < small_block_size; jj++)
                {
                    local_result[ii][jj] = 0;
                }
            }

            const int height = min(block_size_y, size - i);
            const int width = min(block_size_x, size - j);

            for (int k = 0; k < size; k++)
            {
                __shared__ float temp_a[block_size_x], temp_b[block_size_x];
                for (int l = thread_idx; l < height; l += num_thread)
                {
                    temp_a[l] = a[(i + l) * stride + k];
                }
                for (int l = thread_idx; l < width; l += num_thread)
                {
                    temp_b[l] = b[k * stride + (j + l)];
                }
                __syncthreads();

                if (has_result)
                {
                    for (int ii = 0; ii < small_block_size; ii++)
                    {
                        for (int jj = 0; jj < small_block_size; jj++)
                        {
                            local_result[ii][jj] += temp_a[small_block_size * threadIdx.y + ii] *
                                                    temp_b[small_block_size * threadIdx.x + jj];
                        }
                    }
                }
                __syncthreads();
            }
            if (has_result)
            {
                for (int ii = 0; ii < small_block_size; ii++)
                {
                    for (int jj = 0; jj < small_block_size; jj++)
                    {
                        result[(base_i + ii) * stride + (base_j + jj)] = local_result[ii][jj];
                    }
                }
            }
            __syncthreads();
        }
    }
}