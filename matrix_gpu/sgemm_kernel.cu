#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include "sgemm_kernel.cuh"

__global__ void fill(float* data, const int size, const float value)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < size)
    {
        data[idx] = value;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void sgemm(const float* a, const float* b, float* result, const int size, const int stride)
{
    const int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
    const int num_thread = blockDim.x * blockDim.y;
    const int large_ty = threadIdx.y * small_block_size;
    const int large_tx = threadIdx.x * small_block_size;

    // thread 単位で small_balock_size^2 だけ要素を持っている時に確保できる block_size_y
    const int block_size_y = num_thread * small_block_size * small_block_size / block_size_x;
    assert(block_size_y <= block_size_x);

    constexpr int block_k_size = 16;

    for (int i = blockIdx.y * block_size_y; i < size; i += gridDim.y * block_size_y)
    {
        for (int j = blockIdx.x * block_size_x; j < size; j += gridDim.x * block_size_x)
        {
            const bool has_result = (i + large_ty < size && j + large_tx < size);

            // 単一スレッドの結果保存用
            float local_result[small_block_size][small_block_size];
            for (int ii = 0; ii < small_block_size; ii++)
            {
                for (int jj = 0; jj < small_block_size; jj++)
                {
                    local_result[ii][jj] = 0;
                }
            }

            const int height = min(block_size_y, size - i);
            const int width = min(block_size_x, size - j);

            for (int k = 0; k < size; k += block_k_size)
            {
                __shared__ float temp_a[block_k_size][block_size_x + 1], temp_b[block_k_size][block_size_x + 1];
                {
                    const int l = thread_idx % block_size_x;
                    if (l < width)
                    {
                        for (int kk = thread_idx / block_size_x; kk < block_k_size; kk += num_thread / block_size_x)
                        {
                            temp_b[kk][l] = b[(k + kk) * stride + (j + l)];
                        }
                    }
                }
                {
                    const int kk = thread_idx % block_k_size;
                    for (int l = thread_idx / block_k_size; l < height; l += num_thread / block_k_size)
                    {
                        temp_a[kk][l] = a[(i + l) * stride + k + kk];
                    }
                }
                __syncthreads();

                if (has_result)
                {
                    for (int kk = 0; kk < block_k_size; kk++)
                    {
                        for (int ii = 0; ii < small_block_size; ii++)
                        {
                            for (int jj = 0; jj < small_block_size; jj++)
                            {
                                local_result[ii][jj] += temp_a[kk][large_ty + ii] * temp_b[kk][large_tx + jj];
                            }
                        }
                    }
                }
                __syncthreads();
            }
            if (has_result)
            {
                for (int ii = 0; ii < small_block_size; ii++)
                {
                    for (int jj = 0; jj < small_block_size; jj++)
                    {
                        result[(i + large_ty + ii) * stride + (j + large_tx + jj)] = local_result[ii][jj];
                    }
                }
            }
            __syncthreads();
        }
    }
}